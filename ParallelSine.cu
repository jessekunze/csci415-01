#include "hip/hip_runtime.h"
//
// Assignment 1: ParallelSine
// CSCI 415: Networking and Parallel Computation
// Spring 2017
// Name(s): Jesse Kunze
//
// Sine implementation derived from slides here: http://15418.courses.cs.cmu.edu/spring2016/lecture/basicarch


// standard imports
#include <stdio.h>
#include <math.h>
#include <iomanip>
#include <iostream>
#include <string>
#include <time.h>
#include <windows.h>

// problem size (vector length) N
static const int N = 12345678;

// Number of terms to use when approximating sine
static const int TERMS = 6;

// kernel function (CPU - Do not modify)
void sine_serial(float *input, float *output)
{
	int i;

	for (i = 0; i<N; i++) {
		float value = input[i];
		float numer = input[i] * input[i] * input[i];
		int denom = 6; // 3! 
		int sign = -1;
		for (int j = 1; j <= TERMS; j++)
		{
			value += sign * numer / denom;
			numer *= input[i] * input[i];
			denom *= (2 * j + 2) * (2 * j + 3);
			sign *= -1;
		}
		output[i] = value;
	}
}


// kernel function (CUDA device)
// TODO: Implement your graphics kernel here. See assignment instructions for method information
//DONE

__global__ void sine_parallel(float *input, float *output)
{
	int threadId = blockDim.x * blockIdx.x + threadIdx.x;
	if (threadId < N)
	{
		float value = input[threadId];
		float numerator = input[threadId] * input[threadId] * input[threadId];
		int denomenator = 6;
		int sign = -1;
		for (int j = 1; j <= TERMS; j++)
		{
			value += sign * numerator / denomenator;
			numerator *= input[threadId] * input[threadId];
			denomenator *= (2 * j + 2) * (2 * j + 3);
			sign *= -1;
		}
		output[threadId] = value;
	}
}



// BEGIN: timing and error checking routines (do not modify)

int gettimeofday(struct timeval * tp, struct timezone * tzp)
{
	// Taken from http://stackoverflow.com/questions/10905892/equivalent-of-gettimeday-for-windows
	static const uint64_t EPOCH = ((uint64_t)116444736000000000ULL);

	SYSTEMTIME  system_time;
	FILETIME    file_time;
	uint64_t    time;

	GetSystemTime(&system_time);
	SystemTimeToFileTime(&system_time, &file_time);
	time = ((uint64_t)file_time.dwLowDateTime);
	time += ((uint64_t)file_time.dwHighDateTime) << 32;

	tp->tv_sec = (long)((time - EPOCH) / 10000000L);
	tp->tv_usec = (long)(system_time.wMilliseconds * 1000);
	return 0;
}

//Returns the current time in microseconds
long long start_timer() {
	struct timeval tv;
	gettimeofday(&tv, NULL);
	return tv.tv_sec * 1000000 + tv.tv_usec;
}


 //Prints the time elapsed since the specified time
long long stop_timer(long long start_time, std::string name) {
	struct timeval tv;
	gettimeofday(&tv, NULL);
	long long end_time = tv.tv_sec * 1000000 + tv.tv_usec;
	std::cout << std::setprecision(5);
	std::cout << name << ": " << ((float)(end_time - start_time)) / (1000 * 1000) << " sec\n";
	return end_time - start_time;
}


void checkErrors(const char label[])
{
	// we need to synchronise first to catch errors due to
	// asynchroneous operations that would otherwise
	// potentially go unnoticed

	hipError_t err;

	err = hipDeviceSynchronize();
	if (err != hipSuccess)
	{
		char *e = (char*)hipGetErrorString(err);
		fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
	}

	err = hipGetLastError();
	if (err != hipSuccess)
	{
		char *e = (char*)hipGetErrorString(err);
		fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
	}
}

// END: timing and error checking routines (do not modify)



int main(int argc, char **argv)
{
	//BEGIN: CPU implementation (do not modify)
	float *h_cpu_result = (float*)malloc(N*sizeof(float));
	float *h_input = (float*)malloc(N*sizeof(float));
	//Initialize data on CPU
	int i;
	for (i = 0; i<N; i++)
	{
		h_input[i] = 0.1f * i;
	}

	//Execute and time the CPU version
	long long CPU_start_time = start_timer();
	sine_serial(h_input, h_cpu_result);
	long long CPU_time = stop_timer(CPU_start_time, "\nCPU Run Time");
	//END: CPU implementation (do not modify)


	//TODO: Prepare and run your kernel, make sure to copy your results back into h_gpu_result and display your timing results
	float *h_gpu_result = (float*)malloc(N*sizeof(float));

	float * d_input;
	float * d_output;

	long long GPU_start_time = start_timer();

	long long GPU_malloc_start_time = start_timer();
	hipMalloc((void **)&d_input, N*sizeof(float));
	hipMalloc((void **)&d_output, N*sizeof(float));
	long long GPU_memAllocate_time = stop_timer(GPU_malloc_start_time, "\nGPU Memory Allocation Time");

	long long GPU_memToDevice_start_time = start_timer();
	hipMemcpy(d_input, h_input, N*sizeof(float), hipMemcpyHostToDevice);
	long long GPU_memToDevic_time = stop_timer(GPU_memToDevice_start_time, "\nGPU Copy Memory to Device Time");

	long long GPU_kernal_start_time = start_timer();
	sine_parallel << <12057, 1024 >> >(d_input, d_output);
	long long GPU_kernal_time = stop_timer(GPU_kernal_start_time, "\nGPU Kernel Run Time");

	long long GPU_memToHost_start_time = start_timer();
	hipMemcpy(h_gpu_result, d_output, N*sizeof(float), hipMemcpyDeviceToHost);
	long long GPU_memToHost_time = stop_timer(GPU_memToHost_start_time, "\nGPU Copy Memory to Host Time");

	long long GPU_total_run_time = stop_timer(GPU_start_time, "\nGPU Total Run Time");

	// Checking to make sure the CPU and GPU results match - Do not modify
	int errorCount = 0;
	for (i = 0; i<N; i++)
	{
		if (abs(h_cpu_result[i] - h_gpu_result[i]) > 1e-6)
			errorCount = errorCount + 1;
	}
	if (errorCount > 0)
		printf("Result comparison failed.\n");
	else
		printf("Result comparison passed.\n");

	// Cleaning up memory
	free(h_input);
	free(h_cpu_result);
	free(h_gpu_result);
	//addded so the cmd window doesn't close right after running
	system("pause")
	return 0;
}





